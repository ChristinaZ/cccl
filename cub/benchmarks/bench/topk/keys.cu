#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception

#include <cub/device/device_topk.cuh>

#include <nvbench_helper.cuh>

// %RANGE% TUNE_ITEMS_PER_THREAD ipt 4:12:4
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 256:1024:256

#if !TUNE_BASE
template <class KeyInT, class NumItemT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int NOMINAL_4B_ITEMS_PER_THREAD = TUNE_ITEMS_PER_THREAD;
    static constexpr int ITEMS_PER_THREAD = cuda::std::max(1, (NOMINAL_4B_ITEMS_PER_THREAD * 4 / sizeof(KeyInT)));

    static constexpr int BITS_PER_PASS          = hipcub::detail::topk::calc_bits_per_pass<KeyInT>();
    static constexpr int COEFFICIENT_FOR_BUFFER = 128;

    using TopKPolicyT =
      hipcub::AgentTopKPolicy<TUNE_THREADS_PER_BLOCK,
                           ITEMS_PER_THREAD,
                           BITS_PER_PASS,
                           COEFFICIENT_FOR_BUFFER,
                           hipcub::BLOCK_LOAD_VECTORIZE,
                           hipcub::BLOCK_SCAN_WARP_SCANS>;
  };

  using MaxPolicy = policy_t;
};
#endif // !TUNE_BASE

template <typename KeyT, typename NumItemT>
void topk_keys(nvbench::state& state, nvbench::type_list<KeyT, NumItemT>)
{
  using key_input_it_t  = const KeyT*;
  using key_output_it_t = KeyT*;
  using num_items_t     = NumItemT;
  // using select_op_t        = less_then_t<T>;
  // using equality_op_t      = hipcub::NullType;

  constexpr bool select_min       = false;
  constexpr bool is_deterministic = false;
#if !TUNE_BASE
  using policy_t = policy_hub_t<KeyT, NumItemT>;
  using dispatch_t =
    hipcub::DispatchTopK<key_input_it_t,
                      key_output_it_t,
                      hipcub::NullType*,
                      hipcub::NullType*,
                      num_items_t,
                      select_min,
                      is_deterministic,
                      policy_t>;
#else // TUNE_BASE
  using dispatch_t = cub::
    DispatchTopK<key_input_it_t, key_output_it_t, hipcub::NullType*, hipcub::NullType*, num_items_t, select_min, is_deterministic>;
#endif // TUNE_BASE

  // Retrieve axis parameters
  const auto elements          = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  const auto selected_elements = static_cast<std::size_t>(state.get_int64("SelectedElements{io}"));
  const bit_entropy entropy    = str_to_entropy(state.get_string("Entropy"));

  // If possible, do not initialize the input data in the benchmark function.
  // Instead, use the gen function.
  thrust::device_vector<KeyT> in_keys  = generate(elements, entropy);
  thrust::device_vector<KeyT> out_keys = generate(selected_elements);
  key_input_it_t d_keys_in             = thrust::raw_pointer_cast(in_keys.data());
  key_output_it_t d_keys_out           = thrust::raw_pointer_cast(out_keys.data());

  // optionally add memory usage to the state
  //  Calling `state.add_element_count(num_elements)` with the number of input
  //  items will report the item throughput rate in elements-per-second.
  //
  //  Calling `state.add_global_memory_reads<T>(num_elements)` and/or
  //  `state.add_global_memory_writes<T>(num_elements)` will report global device
  //  memory throughput as a percentage of the current device's peak global memory
  //  bandwidth, and also in bytes-per-second.
  //
  //  All of these methods take an optional second `column_name` argument, which
  //  will add a new column to the output with the reported element count / buffer
  //  size and column name.
  state.add_element_count(elements, "NumElements");
  state.add_element_count(selected_elements, "NumSelectedElements");
  state.add_global_memory_reads<KeyT>(elements, "InputKeys");
  state.add_global_memory_writes<KeyT>(selected_elements, "OutputKeys");

  // allocate temporary storage
  std::size_t temp_size;
  dispatch_t::Dispatch(
    nullptr,
    temp_size,
    d_keys_in,
    d_keys_out,
    static_cast<hipcub::NullType*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    elements,
    selected_elements,
    0);
  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  // run the algorithm
  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(
      temp_storage,
      temp_size,
      d_keys_in,
      d_keys_out,
      static_cast<hipcub::NullType*>(nullptr),
      static_cast<hipcub::NullType*>(nullptr),
      elements,
      selected_elements,
      launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(topk_keys, NVBENCH_TYPE_AXES(fundamental_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"KeyT{ct}", "NumItemT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_int64_power_of_two_axis("SelectedElements{io}", nvbench::range(3, 15, 4))
  .add_string_axis("Entropy", {"1.000", "0.544", "0.201", "0.000"});
