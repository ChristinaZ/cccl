/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_topk.cuh>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/memory.h>
#include <thrust/sort.h>

#include <algorithm>

#include "catch2_radix_sort_helper.cuh"
#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"

// %PARAM% TEST_LAUNCH lid 0:1:2
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceTopK::TopKKeys, topk_keys);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceTopK::TopKMinKeys, topk_min_keys);

using key_types       = c2h::type_list<cuda::std::uint32_t, cuda::std::uint64_t>;
using num_items_types = c2h::type_list<cuda::std::uint32_t, cuda::std::uint64_t>;

CUB_TEST("DeviceTopK::TopKKeys: Basic testing", "[keys][topk][device]", key_types, num_items_types)
{
  using key_t       = c2h::get<0, TestType>;
  using num_items_t = c2h::get<1, TestType>;

  // Set input size
  constexpr num_items_t min_num_items = 1 << 10;
  constexpr num_items_t max_num_items = 1 << 15;
  const num_items_t num_items         = GENERATE_COPY(take(5, random(min_num_items, max_num_items)));

  // Set the k value
  constexpr num_items_t min_k = 1 << 3;
  constexpr num_items_t max_k = 1 << 5;
  const num_items_t k         = GENERATE_COPY(take(5, random(min_k, max_k)));

  // Allocate the device memory
  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(k);

  const int num_key_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);

  const bool select_min    = GENERATE(false, true);
  const bool is_descending = !select_min;

  // Run the device-wide API
  if (select_min)
  {
    topk_min_keys(thrust::raw_pointer_cast(in_keys.data()), thrust::raw_pointer_cast(out_keys.data()), num_items, k);
  }
  else
  {
    topk_keys(thrust::raw_pointer_cast(in_keys.data()), thrust::raw_pointer_cast(out_keys.data()), num_items, k);
  }

  // Sort the entire input data as result referece
  c2h::host_vector<key_t> h_in_keys(in_keys);
  c2h::host_vector<key_t> host_results;
  host_results.resize(out_keys.size());
  if (is_descending)
  {
    std::partial_sort_copy(
      h_in_keys.begin(), h_in_keys.end(), host_results.begin(), host_results.end(), std::greater<key_t>());
  }
  else
  {
    std::partial_sort_copy(
      h_in_keys.begin(), h_in_keys.end(), host_results.begin(), host_results.end(), std::less<key_t>());
  }
  // Since the results of API TopKMinKeys() and TopKKeys() are not-sorted
  // We need to sort the results first.
  c2h::host_vector<key_t> device_results(out_keys);
  if (is_descending)
  {
    std::stable_sort(device_results.begin(), device_results.end(), std::greater<key_t>());
  }
  else
  {
    std::stable_sort(device_results.begin(), device_results.end(), std::less<key_t>());
  }

  REQUIRE(host_results == device_results);
}
