/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_topk.cuh>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/memory.h>

#include <algorithm>

#include "catch2_radix_sort_helper.cuh"
#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"

// %PARAM% TEST_LAUNCH lid 0:1:2
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceTopK::TopKPairs, topk_pairs);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceTopK::TopKMinPairs, topk_min_pairs);

template <typename key_t, typename value_t>
struct comparator_t
{
  const key_t* key_arr;
  const value_t* value_arr;
  bool is_descending{};

  comparator_t(key_t* key_arr, value_t* value_arr, bool is_descending)
      : key_arr(key_arr)
      , value_arr(value_arr)
      , is_descending(is_descending)
  {}

  bool operator()(std::size_t a, std::size_t b)
  {
    bool res;
    if (is_descending)
    {
      if (key_arr[a] > key_arr[b])
      {
        res = true;
      }
      else if ((key_arr[a] == key_arr[b]) && (value_arr[a] < value_arr[b]))
      {
        res = true;
      }
      else
      {
        res = false;
      }
    }
    else
    {
      if (key_arr[a] < key_arr[b])
      {
        res = true;
      }
      else if ((key_arr[a] == key_arr[b]) && (value_arr[a] < value_arr[a]))
      {
        res = true;
      }
      else
      {
        res = false;
      }
    }

    return res;
  }
};

template <typename key_t, typename value_t>
void sort_keys_and_values(c2h::device_vector<key_t>& keys,
                          c2h::device_vector<value_t>& values,
                          std::pair<c2h::host_vector<key_t>, c2h::host_vector<value_t>>& results,
                          bool is_descending)
{
  c2h::host_vector<key_t> h_keys(keys);
  c2h::host_vector<value_t> h_values(values);

  c2h::host_vector<std::size_t> h_permutation(keys.size());
  thrust::sequence(h_permutation.begin(), h_permutation.end());
  comparator_t<key_t, value_t> comp{
    thrust::raw_pointer_cast(h_keys.data()), thrust::raw_pointer_cast(h_values.data()), is_descending};
  std::stable_sort(h_permutation.begin(), h_permutation.end(), comp);

  thrust::gather(h_permutation.cbegin(),
                 h_permutation.cend(),
                 thrust::make_zip_iterator(h_keys.cbegin(), h_values.cbegin()),
                 thrust::make_zip_iterator(results.first.begin(), results.second.begin()));
}

using value_types     = c2h::type_list<cuda::std::uint32_t, cuda::std::uint64_t>;
using num_items_types = c2h::type_list<cuda::std::uint32_t, cuda::std::uint64_t>;

CUB_TEST("DeviceTopK::TopKPairs: Basic testing", "[pairs][topk][device]", value_types, num_items_types)
{
  using key_t       = cuda::std::uint32_t;
  using value_t     = c2h::get<0, TestType>;
  using num_items_t = c2h::get<1, TestType>;

  // Set input size
  constexpr num_items_t min_num_items = 1 << 10;
  constexpr num_items_t max_num_items = 1 << 15;
  const num_items_t num_items         = GENERATE_COPY(take(5, random(min_num_items, max_num_items)));

  // Set the k value
  constexpr num_items_t min_k = 1 << 3;
  constexpr num_items_t max_k = 1 << 5;
  const num_items_t k         = GENERATE_COPY(take(5, random(min_k, max_k)));

  // Allocate the device memory
  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(k);

  c2h::device_vector<value_t> in_values(num_items);
  c2h::device_vector<value_t> out_values(k);

  const int num_key_seeds   = 1;
  const int num_value_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);
  c2h::gen(CUB_SEED(num_value_seeds), in_values);

  const bool select_min    = GENERATE(false, true);
  const bool is_descending = !select_min;

  // Run the device-wide API
  if (select_min)
  {
    topk_min_pairs(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      num_items,
      k);
  }
  else
  {
    topk_pairs(thrust::raw_pointer_cast(in_keys.data()),
               thrust::raw_pointer_cast(out_keys.data()),
               thrust::raw_pointer_cast(in_values.data()),
               thrust::raw_pointer_cast(out_values.data()),
               num_items,
               k);
  }

  // Sort the entire input data as result referece
  std::pair<c2h::host_vector<key_t>, c2h::host_vector<value_t>> in_results;
  in_results.first.resize(in_keys.size());
  in_results.second.resize(in_keys.size());
  sort_keys_and_values(in_keys, in_values, in_results, is_descending);

  // Since the results of API TopKMinPairs() and TopKPairs() are not-sorted
  // We need to sort the results first.
  std::pair<c2h::host_vector<key_t>, c2h::host_vector<value_t>> out_results;
  out_results.first.resize(out_keys.size());
  out_results.second.resize(out_keys.size());
  sort_keys_and_values(out_keys, out_values, out_results, is_descending);

  // i for results from gpu (TopKMinPairs() and TopKPairs()); j for reference results
  num_items_t i = 0, j = 0;
  bool res = true;
  while (i < k && j < num_items)
  {
    if (out_results.first[i] == in_results.first[j])
    {
      if (out_results.second[i] == in_results.second[j])
      {
        i++;
        j++;
      }
      else if (out_results.second[i] > in_results.second[j])
      {
        // Since the results of API TopKMinPairs() and TopKPairs() are not stable.
        // There might be multiple items equaling to the value of kth element,
        // any of them can appear in the results. We need to find them from the input data.
        j++;
      }
      else
      {
        res = false;
        break;
      }
    }
    else
    {
      res = false;
      break;
    }
  }
  REQUIRE(res == true);
}
