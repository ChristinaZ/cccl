/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_topk.cuh>
#include <cub/util_type.cuh>

#include <thrust/memory.h>

#include <cuda/std/type_traits>

#include <algorithm>
#include <cstdint>
#include <limits>
#include <new> // bad_alloc

#include "catch2_large_array_sort_helper.cuh"
#include "catch2_radix_sort_helper.cuh"
#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceTopK::TopKPairs, topk_pairs);

using custom_value_t = c2h::custom_type_t<c2h::equal_comparable_t>;
// using value_types    = c2h::type_list<cuda::std::uint32_t, cuda::std::uint64_t>;

// hipcub::detail::ChooseOffsetsT only selected 32/64 bit unsigned types:
// using num_items_types = c2h::type_list<cuda::std::uint32_t, cuda::std::uint64_t>;
using value_types     = c2h::type_list<cuda::std::uint32_t>;
using num_items_types = c2h::type_list<cuda::std::uint32_t>;

CUB_TEST("DeviceTopK::TopKPairs: Basic testing", "[pairs][topk][device]", value_types, num_items_types)
{
  using key_t       = cuda::std::uint32_t;
  using value_t     = c2h::get<0, TestType>;
  using num_items_t = c2h::get<1, TestType>;

  constexpr num_items_t min_num_items = 1 << 5;
  constexpr num_items_t max_num_items = 1 << 20;
  const num_items_t num_items =
    GENERATE_COPY(num_items_t{0}, num_items_t{1}, take(5, random(min_num_items, max_num_items)));

  constexpr num_items_t min_k = 1 << 3;
  constexpr num_items_t max_k = 1 << 15;
  const num_items_t k         = GENERATE_COPY(num_items_t{0}, num_items_t{1}, take(5, random(min_k, max_k)));

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);

  c2h::device_vector<value_t> in_values(k);
  c2h::device_vector<value_t> out_values(k);

  const int num_key_seeds   = 1;
  const int num_value_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);
  c2h::gen(CUB_SEED(num_value_seeds), in_values);

  const bool is_descending = GENERATE(false, true);

  if (is_descending)
  {
    // sort_pairs_descending(
    //   thrust::raw_pointer_cast(in_keys.data()),
    //   thrust::raw_pointer_cast(out_keys.data()),
    //   thrust::raw_pointer_cast(in_values.data()),
    //   thrust::raw_pointer_cast(out_values.data()),
    //   num_items,
    //   begin_bit<key_t>(),
    //   end_bit<key_t>());
  }
  else
  {
    topk_pairs(thrust::raw_pointer_cast(in_keys.data()),
               thrust::raw_pointer_cast(out_keys.data()),
               thrust::raw_pointer_cast(in_values.data()),
               thrust::raw_pointer_cast(out_values.data()),
               num_items,
               k);
  }

  auto refs        = radix_sort_reference(in_keys, in_values, is_descending);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  int num_equaling_kth   = 0;
  num_items_t kth_offset = k - 1;
  key_t kth_element      = ref_keys[kth_offset];
  key_t pre_kth_element  = kth_element;
  while (pre_kth_element == kth_element && kth_offset != 0)
  {
    num_equaling_kth++;
    pre_kth_element = ref_keys[kth_offset];
    kth_offset--;
  }

  c2h::device_vector<value_t> out_values_unde(out_values.begin(), out_values.end() - num_equaling_kth);
  c2h::host_vector<value_t> ref_values_unde(ref_values.begin(), ref_values.end() - num_equaling_kth);

  REQUIRE(ref_keys == out_keys);
  REQUIRE(ref_values_unde == out_values_unde);
}
